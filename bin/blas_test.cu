#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <test_util.h>
#include <face_quda.h>

// include because of nasty globals used in the tests
#include <dslash_util.h>

// Wilson, clover-improved Wilson, and twisted mass are supported.
extern QudaDslashType dslash_type;
extern bool tune;
extern int device;
extern int xdim;
extern int ydim;
extern int zdim;
extern int tdim;
extern int gridsize_from_cmdline[];
extern int niter;

extern bool tune;

extern void usage(char** );

const int Nkernels = 32;

using namespace quda;

cpuColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *lH;
cudaColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *lD;
int Nspin;

void setPrec(ColorSpinorParam &param, const QudaPrecision precision)
{
  param.precision = precision;
  if (Nspin == 1 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void
display_test_info()
{
  printfQuda("running the following test:\n");
    
  printfQuda("S_dimension T_dimension Nspin\n");
  printfQuda("%d/%d/%d        %d      %d\n", xdim, ydim, zdim, tdim, Nspin);     

  printfQuda("Grid partition info:     X  Y  Z  T\n"); 
  printfQuda("                         %d  %d  %d  %d\n", 
	     commDimPartitioned(0),
	     commDimPartitioned(1),
	     commDimPartitioned(2),
	     commDimPartitioned(3)); 
  
  return;  
}

void initFields(int prec)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec;
  QudaPrecision low_aux_prec;

  ColorSpinorParam param;
  param.nColor = 3;
  // set spin according to the type of dslash
  Nspin = (dslash_type == QUDA_ASQTAD_DSLASH) ? 1 : 4;
  param.nSpin = Nspin;
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields
  param.siteSubset = QUDA_PARITY_SITE_SUBSET;
  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = xdim/2;
  else param.x[0] = xdim;
  param.x[1] = ydim;
  param.x[2] = zdim;
  param.x[3] = tdim;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.precision = QUDA_DOUBLE_PRECISION;
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

  vH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  wH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  xH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  yH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  zH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  hH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  lH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);

  // Now set the parameters for the cuda fields
  //param.pad = xdim*ydim*zdim/2;
  
  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_HALF_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_SINGLE_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_SINGLE_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_DOUBLE_PRECISION);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  }

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  *vD = *vH;
  *wD = *wH;
  *xD = *xH;
  *yD = *yH;
  *zD = *zH;
  *hD = *hH;
  *lD = *lH;
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete lD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete lH;
}


double benchmark(int kernel, const int niter) {

  double a, b, c;
  quda::Complex a2, b2, c2;

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  for (int i=0; i < niter; ++i) {

    switch (kernel) {

    case 0:
      copyCuda(*yD, *hD);
      break;

    case 1:
      copyCuda(*yD, *lD);
      break;
      
    case 2:
      axpbyCuda(a, *xD, b, *yD);
      break;

    case 3:
      xpyCuda(*xD, *yD);
      break;

    case 4:
      axpyCuda(a, *xD, *yD);
      break;

    case 5:
      xpayCuda(*xD, a, *yD);
      break;

    case 6:
      mxpyCuda(*xD, *yD);
      break;

    case 7:
      axCuda(a, *xD);
      break;

    case 8:
      caxpyCuda(a2, *xD, *yD);
      break;

    case 9:
      caxpbyCuda(a2, *xD, b2, *yD);
      break;

    case 10:
      cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
      break;

    case 11:
      axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
      break;

    case 12:
      axpyZpbxCuda(a, *xD, *yD, *zD, b);
      break;

    case 13:
      caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
      break;
      
    case 14:
      cabxpyAxCuda(a, b2, *xD, *yD);
      break;

    case 15:
      caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      break;

    case 16:
      caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      break;

    case 17:
      caxpyXmazCuda(a2, *xD, *yD, *zD);
      break;

      // double
    case 18:
      normCuda(*xD);
      break;

    case 19:
      reDotProductCuda(*xD, *yD);
      break;

    case 20:
      axpyNormCuda(a, *xD, *yD);
      break;

    case 21:
      xmyNormCuda(*xD, *yD);
      break;
      
    case 22:
      caxpyNormCuda(a2, *xD, *yD);
      break;

    case 23:
      caxpyXmazNormXCuda(a2, *xD, *yD, *zD);
      break;

    case 24:
      cabxpyAxNormCuda(a, b2, *xD, *yD);
      break;

    // double2
    case 25:
      cDotProductCuda(*xD, *yD);
      break;

    case 26:
      xpaycDotzyCuda(*xD, a, *yD, *zD);
      break;
      
    case 27:
      caxpyDotzyCuda(a2, *xD, *yD, *zD);
      break;

    // double3
    case 28:
      cDotProductNormACuda(*xD, *yD);
      break;

    case 29:
      cDotProductNormBCuda(*xD, *yD);
      break;

    case 30:
      caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    case 31:
      HeavyQuarkResidualNormCuda(*xD, *yD);
      break;

    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }
  
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);

  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(norm2(*a##D) - norm2(*a##H)) / norm2(*a##H)

double test(int kernel) {

  double a = 1.5, b = 2.5, c = 3.5;
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;

  switch (kernel) {

  case 0:
    *hD = *hH;
    copyCuda(*yD, *hD);
    yH->copy(*hH);
    error = ERROR(y);
    break;

  case 1:
    *lD = *lH;
    copyCuda(*yD, *lD);
    yH->copy(*lH);
    error = ERROR(y);
    break;
      
  case 2:
    *xD = *xH;
    *yD = *yH;
    axpbyCuda(a, *xD, b, *yD);
    axpbyCpu(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    xpyCuda(*xD, *yD);
    xpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    axpyCuda(a, *xD, *yD);
    axpyCpu(a, *xH, *yH);
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    xpayCuda(*xD, a, *yD);
    xpayCpu(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    mxpyCuda(*xD, *yD);
    mxpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    axCuda(a, *xD);
    axCpu(a, *xH);
    error = ERROR(x);
    break;

  case 8:
    *xD = *xH;
    *yD = *yH;
    caxpyCuda(a2, *xD, *yD);
    caxpyCpu(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    caxpbyCuda(a2, *xD, b2, *yD);
    caxpbyCpu(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
    cxpaypbzCpu(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
    axpyBzpcxCpu(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyZpbxCuda(a, *xD, *yD, *zD, b);
    axpyZpbxCpu(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
    caxpbypzYmbwCpu(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;
      
  case 14:
    *xD = *xH;
    *yD = *yH;
    cabxpyAxCuda(a, b2, *xD, *yD);
    cabxpyAxCpu(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      caxpbypzCpu(a2, *xH, b2, *yH, *zH);
      error = ERROR(z); }
    break;
    
  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    {caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      caxpbypczpwCpu(a2, *xH, b2, *yH, c2, *zH, *wH);
      error = ERROR(w); }
    break;

  case 17:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpyXmazCuda(a, *xD, *yD, *zD);
     caxpyXmazCpu(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

    // double
  case 18:
    *xD = *xH;
    error = fabs(normCuda(*xD) - normCpu(*xH)) / normCpu(*xH);
    break;
    
  case 19:
    *xD = *xH;
    *yD = *yH;
    error = fabs(reDotProductCuda(*xD, *yD) - reDotProductCpu(*xH, *yH)) / fabs(reDotProductCpu(*xH, *yH));
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = axpyNormCuda(a, *xD, *yD);
    double h = axpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = xmyNormCuda(*xD, *yD);
    double h = xmyNormCpu(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;
    
  case 22:
    *xD = *xH;
    *yD = *yH;
    {double d = caxpyNormCuda(a, *xD, *yD);
    double h = caxpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = caxpyXmazNormXCuda(a, *xD, *yD, *zD);
      double h = caxpyXmazNormXCpu(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    {double d = cabxpyAxNormCuda(a, b2, *xD, *yD);
      double h = cabxpyAxNormCpu(a, b2, *xH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

    // double2
  case 25:
    *xD = *xH;
    *yD = *yH;
    error = abs(cDotProductCuda(*xD, *yD) - cDotProductCpu(*xH, *yH)) / abs(cDotProductCpu(*xH, *yH));
    break;
    
  case 26:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { quda::Complex d = xpaycDotzyCuda(*xD, a, *yD, *zD);
      quda::Complex h = xpaycDotzyCpu(*xH, a, *yH, *zH);
      error =  fabs(norm2(*yD) - norm2(*yH)) / norm2(*yH) + abs(d-h)/abs(h);
    }
    break;
    
  case 27:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = caxpyDotzyCuda(a, *xD, *yD, *zD);
      quda::Complex h = caxpyDotzyCpu(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

    // double3
  case 28:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormACuda(*xD, *yD);
      double3 h = cDotProductNormACpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormBCuda(*xD, *yD);
      double3 h = cDotProductNormBCpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = caxpbypzYmbwcDotProductUYNormYCpu(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  case 31:
    *xD = *xH;
    *yD = *yH;
    { double3 d = HeavyQuarkResidualNormCuda(*xD, *yD);
      double3 h = HeavyQuarkResidualNormCpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }

  return error;
}

int main(int argc, char** argv)
{
  for (int i = 1; i < argc; i++){
    if(process_command_line_option(argc, argv, &i) == 0){
      continue;
    } 
    printfQuda("ERROR: Invalid option:%s\n", argv[i]);
    usage(argv);
  }

  setSpinorSiteSize(24);
  initCommsQuda(argc, argv, gridsize_from_cmdline, 4);
  display_test_info();
  initQuda(device);

  char *names[] = {
    "copyHS",
    "copyLS",
    "axpby",
    "xpy",
    "axpy",
    "xpay",
    "mxpy",
    "ax",
    "caxpy",
    "caxpby",
    "cxpaypbz",
    "axpyBzpcx",
    "axpyZpbx",
    "caxpbypzYmbw",
    "cabxpyAx",
    "caxpbypz",
    "caxpbypczpw",
    "caxpyXmaz",
    "norm",
    "reDotProduct",
    "axpyNorm",
    "xmyNorm",
    "caxpyNorm",
    "caxpyXmazNormX",
    "cabxpyAxNorm",
    "cDotProduct",
    "xpaycDotzy",
    "caxpyDotzy",
    "cDotProductNormA",
    "cDotProductNormB",
    "caxpbypzYmbwcDotProductWYNormY",
    "HeavyQuarkResidualNorm"
  };

  char *prec_str[] = {"half", "single", "double"};
  
  // Only benchmark double precision if supported
#if (__COMPUTE_CAPABILITY__ >= 130)
  int Nprec = 3;
#else
  int Nprec = 2;
#endif

  // enable the tuning
  quda::setBlasTuning(tune ? QUDA_TUNE_YES : QUDA_TUNE_NO, QUDA_SILENT);

  for (int prec = 0; prec < Nprec; prec++) {

    printfQuda("\nBenchmarking %s precision with %d iterations...\n\n", prec_str[prec], niter);
    initFields(prec);

    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;

      // do the initial tune
      benchmark(kernel, 1);
    
      // now rerun with more iterations to get accurate speed measurements
      quda::blas_flops = 0;
      quda::blas_bytes = 0;
      
      double secs = benchmark(kernel, niter);
      
      double gflops = (quda::blas_flops*1e-9)/(secs);
      double gbytes = quda::blas_bytes/(secs*1e9);
    
      printfQuda("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
    }
    freeFields();
  }

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  for (int prec = 0; prec < Nprec; prec++) {
    printfQuda("\nTesting %s precision...\n\n", prec_str[prec]);
    initFields(prec);
    
    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;
      double error = test(kernel);
      printfQuda("%-35s error = %e, \n", names[kernel], error);
    }
    freeFields();
  }

  endQuda();

  endCommsQuda();
}
