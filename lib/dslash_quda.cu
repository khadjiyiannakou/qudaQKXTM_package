#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC


#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

enum KernelType {
  INTERIOR_KERNEL = 5,
  EXTERIOR_KERNEL_X = 0,
  EXTERIOR_KERNEL_Y = 1,
  EXTERIOR_KERNEL_Z = 2,
  EXTERIOR_KERNEL_T = 3
};

namespace quda {

  struct DslashParam {
    int threads; // the desired number of active threads
    int parity;  // Even-Odd or Odd-Even
    int commDim[QUDA_MAX_DIM]; // Whether to do comms or not
    int ghostDim[QUDA_MAX_DIM]; // Whether a ghost zone has been allocated for a given dimension
    int ghostOffset[QUDA_MAX_DIM];
    int ghostNormOffset[QUDA_MAX_DIM];
    KernelType kernel_type; //is it INTERIOR_KERNEL, EXTERIOR_KERNEL_X/Y/Z/T
  };

  DslashParam dslashParam;

  // these are set in initDslashConst
  int Vspatial;

  static hipEvent_t packEnd[Nstream];
  static hipEvent_t gatherStart[Nstream];
  static hipEvent_t gatherEnd[Nstream];
  static hipEvent_t scatterStart[Nstream];
  static hipEvent_t scatterEnd[Nstream];

  static struct timeval dslashStart_h;
#ifdef MULTI_GPU
  static struct timeval commsStart[Nstream];
  static struct timeval commsEnd[Nstream];
#endif

  // these events are only used for profiling
#ifdef DSLASH_PROFILING
#define DSLASH_TIME_PROFILE() dslashTimeProfile()

  static hipEvent_t dslashStart;
  static hipEvent_t dslashEnd;
  static hipEvent_t packStart[Nstream];
  static hipEvent_t kernelStart[Nstream];
  static hipEvent_t kernelEnd[Nstream];

  // dimension 2 because we want absolute and relative
  float packTime[Nstream][2];
  float gatherTime[Nstream][2];
  float commsTime[Nstream][2];
  float scatterTime[Nstream][2];
  float kernelTime[Nstream][2];
  float dslashTime;
#define CUDA_EVENT_RECORD(a,b) hipEventRecord(a,b)
#else
#define CUDA_EVENT_RECORD(a,b)
#define DSLASH_TIME_PROFILE()
#endif

  static FaceBuffer *face;
  static cudaColorSpinorField *inSpinor;

  // For tuneLaunch() to uniquely identify a suitable set of launch parameters, we need copies of a few of
  // the constants set by initDslashConstants().
  static struct {
    int x[4];
    int Ls;
    unsigned long long VolumeCB() { return x[0]*x[1]*x[2]*x[3]/2; }
    // In the future, we may also want to add gauge_fixed, sp_stride, ga_stride, cl_stride, etc.
  } dslashConstants;

  // dslashTuning = QUDA_TUNE_YES enables autotuning when the dslash is
  // first launched
  static QudaTune dslashTuning = QUDA_TUNE_NO;
  static QudaVerbosity verbosity = QUDA_SILENT;

  void setDslashTuning(QudaTune tune, QudaVerbosity verbose)
  {
    dslashTuning = tune;
    verbosity = verbose;
  }

  // determines whether the temporal ghost zones are packed with a gather kernel,
  // as opposed to multiple calls to hipMemcpy()
  static bool kernelPackT = false;

  void setKernelPackT(bool packT) { kernelPackT = packT; }

  bool getKernelPackT() { return kernelPackT; }


#include <dslash_textures.h>
#include <dslash_constants.h>

#if defined(DIRECT_ACCESS_LINK) || defined(DIRECT_ACCESS_WILSON_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_ACCUM) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_INTER) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR) || \
  defined(DIRECT_ACCESS_CLOVER)

  static inline __device__ float short2float(short a) {
    return (float)a/MAX_SHORT;
  }

  static inline __device__ short float2short(float c, float a) {
    return (short)(a*c*MAX_SHORT);
  }

  static inline __device__ short4 float42short4(float c, float4 a) {
    return make_short4(float2short(c, a.x), float2short(c, a.y), float2short(c, a.z), float2short(c, a.w));
  }

  static inline __device__ float4 short42float4(short4 a) {
    return make_float4(short2float(a.x), short2float(a.y), short2float(a.z), short2float(a.w));
  }

  static inline __device__ float2 short22float2(short2 a) {
    return make_float2(short2float(a.x), short2float(a.y));
  }
#endif // DIRECT_ACCESS inclusions

  // Enable shared memory dslash for Fermi architecture
  //#define SHARED_WILSON_DSLASH
  //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#include <pack_face_def.h>        // kernels for packing the ghost zones and general indexing
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <tm_core.h>              // solo twisted mass kernel


#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif



  void setFace(const FaceBuffer &Face) {
    face = (FaceBuffer*)&Face; // nasty
  }


  void createDslashEvents()
  {
#ifndef DSLASH_PROFILING
    // add hipEventDisableTiming for lower sync overhead
    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packEnd[i], hipEventDisableTiming);
      hipEventCreate(&gatherStart[i], hipEventDisableTiming);
      hipEventCreate(&gatherEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterEnd[i], hipEventDisableTiming);
    }
#else
    hipEventCreate(&dslashStart);
    hipEventCreate(&dslashEnd);
    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packStart[i]);
      hipEventCreate(&packEnd[i]);

      hipEventCreate(&gatherStart[i]);
      hipEventCreate(&gatherEnd[i]);

      hipEventCreate(&scatterStart[i]);
      hipEventCreate(&scatterEnd[i]);

      hipEventCreate(&kernelStart[i]);
      hipEventCreate(&kernelEnd[i]);

      kernelTime[i][0] = 0.0;
      kernelTime[i][1] = 0.0;

      gatherTime[i][0] = 0.0;
      gatherTime[i][1] = 0.0;

      commsTime[i][0] = 0.0;
      commsTime[i][1] = 0.0;

      scatterTime[i][0] = 0.0;
      scatterTime[i][1] = 0.0;
    }
#endif

    checkCudaError();
  }


  void destroyDslashEvents()
  {
    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packEnd[i]);
      hipEventDestroy(gatherStart[i]);
      hipEventDestroy(gatherEnd[i]);
      hipEventDestroy(scatterStart[i]);
      hipEventDestroy(scatterEnd[i]);
    }

#ifdef DSLASH_PROFILING
    hipEventDestroy(dslashStart);
    hipEventDestroy(dslashEnd);

    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packStart[i]);
      hipEventDestroy(kernelStart[i]);
      hipEventDestroy(kernelEnd[i]);
    }
#endif

    checkCudaError();
  }


#define MORE_GENERIC_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...) \
  if (x==0) {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else {								\
      FUNC ## 8 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  } else {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
      FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  }

#ifndef MULTI_GPU

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  default:								\
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
  }

#else

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_X:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Y:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Z:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_T:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  }

#endif

  // macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...)	\
  if (!dagger) {							\
    GENERIC_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      } else {								\
    GENERIC_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      }

  // macro used for staggered dslash
#define STAGGERED_DSLASH(gridDim, blockDim, shared, stream, param, ...)	\
  GENERIC_DSLASH(staggeredDslash, , Axpy, gridDim, blockDim, shared, stream, param, __VA_ARGS__)


#define MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...) \
  if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
    FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
    FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
    FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  }									

#ifndef MULTI_GPU

#define GENERIC_ASYM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  default:								\
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
  }

#else

#define GENERIC_ASYM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_X:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Y:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Z:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_T:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  }

#endif

  // macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define ASYM_DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...) \
  if (!dagger) {							\
    GENERIC_ASYM_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      } else {								\
    GENERIC_ASYM_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      }


  // Use an abstract class interface to drive the different CUDA dslash
  // kernels. All parameters are curried into the derived classes to
  // allow a simple interface.
  class DslashCuda : public Tunable {
  protected:
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool advanceGridDim(TuneParam &param) const { return false; } // Don't tune the grid dimensions.
    bool advanceBlockDim(TuneParam &param) const {
      bool advance = Tunable::advanceBlockDim(param);
      if (advance) param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
      return advance;
    }

  public:
    DslashCuda() { }
    virtual ~DslashCuda() { }
    virtual TuneKey tuneKey() const;
    std::string paramString(const TuneParam &param) const // Don't bother printing the grid dim.
    {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }
    virtual int Nface() { return 2; }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
    }


  };

  TuneKey DslashCuda::tuneKey() const
  {
    std::stringstream vol, aux;
  
    vol << dslashConstants.x[0] << "x";
    vol << dslashConstants.x[1] << "x";
    vol << dslashConstants.x[2] << "x";
    vol << dslashConstants.x[3];

    aux << "type=";
#ifdef MULTI_GPU
    char comm[5], ghost[5];
    switch (dslashParam.kernel_type) {
    case INTERIOR_KERNEL: aux << "interior"; break;
    case EXTERIOR_KERNEL_X: aux << "exterior_x"; break;
    case EXTERIOR_KERNEL_Y: aux << "exterior_y"; break;
    case EXTERIOR_KERNEL_Z: aux << "exterior_z"; break;
    case EXTERIOR_KERNEL_T: aux << "exterior_t"; break;
    }
    for (int i=0; i<4; i++) {
      comm[i] = (dslashParam.commDim[i] ? '1' : '0');
      ghost[i] = (dslashParam.ghostDim[i] ? '1' : '0');
    }
    comm[4] = '\0'; ghost[4] = '\0';
    aux << ",comm=" << comm;
    if (dslashParam.kernel_type == INTERIOR_KERNEL) {
      aux << ",ghost=" << ghost;
    }
#else
    aux << "single-GPU";
#endif // MULTI_GPU
    return TuneKey(vol.str(), typeid(*this).name(), aux.str());
  }

  /** This derived class is specifically for driving the Dslash kernels
      that use shared memory blocking.  This only applies on Fermi and
      upwards, and only for the interior kernels. */
#if (__COMPUTE_CAPABILITY__ >= 200 && defined(SHARED_WILSON_DSLASH)) 
  class SharedDslashCuda : public DslashCuda {
  protected:
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; } // FIXME: this isn't quite true, but works
    bool advanceSharedBytes(TuneParam &param) const { 
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::advanceSharedBytes(param);
      else return false;
    } // FIXME - shared memory tuning only supported on exterior kernels

    /** Helper function to set the shared memory size from the 3-d block size */
    int sharedBytes(const dim3 &block) const { 
      int warpSize = 32; // FIXME - query from device properties
      int block_xy = block.x*block.y;
      if (block_xy % warpSize != 0) block_xy = ((block_xy / warpSize) + 1)*warpSize;
      return block_xy*block.z*sharedBytesPerThread();
    }

    /** Helper function to set the 3-d grid size from the 3-d block size */
    dim3 createGrid(const dim3 &block) const {
      unsigned int gx = ((dslashConstants.x[0]/2)*dslashConstants.x[3] + block.x - 1) / block.x;
      unsigned int gy = (dslashConstants.x[1] + block.y - 1 ) / block.y;	
      unsigned int gz = (dslashConstants.x[2] + block.z - 1) / block.z;
      return dim3(gx, gy, gz);
    }

    /** Advance the 3-d block size. */
    bool advanceBlockDim(TuneParam &param) const {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::advanceBlockDim(param);
      const unsigned int min_threads = 2;
      const unsigned int max_threads = 512; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int max_shared = 16384*3; // FIXME: use deviceProp.sharedMemPerBlock;
    
      // set the x-block dimension equal to the entire x dimension
      bool set = false;
      dim3 blockInit = param.block;
      blockInit.z++;
      for (unsigned bx=blockInit.x; bx<=dslashConstants.x[0]/2; bx++) {
	//unsigned int gx = (dslashConstants.x[0]*dslashConstants.x[3] + bx - 1) / bx;
	for (unsigned by=blockInit.y; by<=dslashConstants.x[1]; by++) {
	  unsigned int gy = (dslashConstants.x[1] + by - 1 ) / by;	
	
	  if (by > 1 && (by%2) != 0) continue; // can't handle odd blocks yet except by=1
	
	  for (unsigned bz=blockInit.z; bz<=dslashConstants.x[2]; bz++) {
	    unsigned int gz = (dslashConstants.x[2] + bz - 1) / bz;
	  
	    if (bz > 1 && (bz%2) != 0) continue; // can't handle odd blocks yet except bz=1
	    if (bx*by*bz > max_threads) continue;
	    if (bx*by*bz < min_threads) continue;
	    // can't yet handle the last block properly in shared memory addressing
	    if (by*gy != dslashConstants.x[1]) continue;
	    if (bz*gz != dslashConstants.x[2]) continue;
	    if (sharedBytes(dim3(bx, by, bz)) > max_shared) continue;

	    param.block = dim3(bx, by, bz);	  
	    set = true; break;
	  }
	  if (set) break;
	  blockInit.z = 1;
	}
	if (set) break;
	blockInit.y = 1;
      }

      if (param.block.x > dslashConstants.x[0]/2 && param.block.y > dslashConstants.x[1] &&
	  param.block.z > dslashConstants.x[2] || !set) {
	//||sharedBytesPerThread()*param.block.x > max_shared) {
	param.block = dim3(dslashConstants.x[0]/2, 1, 1);
	return false;
      } else { 
	param.grid = createGrid(param.block);
	param.shared_bytes = sharedBytes(param.block);
	return true; 
      }
    
    }

  public:
    SharedDslashCuda() : DslashCuda() { ; }
    virtual ~SharedDslashCuda() { ; }
    std::string paramString(const TuneParam &param) const // override and print out grid as well
    {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "grid=(" << param.grid.x << "," << param.grid.y << "," << param.grid.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::initTuneParam(param);

      param.block = dim3(dslashConstants.x[0]/2, 1, 1);
      param.grid = createGrid(param.block);
      param.shared_bytes = sharedBytes(param.block);
    }

    /** Sets default values for when tuning is disabled - this is guaranteed to work, but will be slow */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) DslashCuda::defaultTuneParam(param);
      else initTuneParam(param);
    }
  };
#else /** For pre-Fermi architectures */
  class SharedDslashCuda : public DslashCuda {
  public:
    SharedDslashCuda() : DslashCuda() { }
    virtual ~SharedDslashCuda() { }
  };
#endif


  template <typename sFloat, typename gFloat>
  class WilsonDslashCuda : public SharedDslashCuda {

  private:
    const size_t bytes, norm_bytes;
    sFloat *out;
    float *outNorm;
    char *saveOut, *saveOutNorm;
    const sFloat *in, *x;
    const float *inNorm, *xNorm;
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const int dagger;
    const double a;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200) // Fermi uses shared memory for common input
      if (dslashParam.kernel_type == INTERIOR_KERNEL) { // Interior kernels use shared memory for common iunput
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else { // Exterior kernels use no shared memory
	return 0;
      }
#else // Pre-Fermi uses shared memory only for pseudo-registers
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }

  public:
    WilsonDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		     const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		     const sFloat *x, const float *xNorm, const double a,
		     const int dagger, const size_t bytes, const size_t norm_bytes)
      : SharedDslashCuda(), bytes(bytes), norm_bytes(norm_bytes), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), in(in), 
	inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a)
    { 
      bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
    }

    virtual ~WilsonDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      DSLASH(dslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     out, outNorm, gauge0, gauge1, in, inNorm, x, xNorm, a);
    }

    void preTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	saveOut = new char[bytes];
	hipMemcpy(saveOut, out, bytes, hipMemcpyDeviceToHost);
	if (typeid(sFloat) == typeid(short4)) {
	  saveOutNorm = new char[norm_bytes];
	  hipMemcpy(saveOutNorm, outNorm, norm_bytes, hipMemcpyDeviceToHost);
	}
      }
    }

    void postTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	hipMemcpy(out, saveOut, bytes, hipMemcpyHostToDevice);
	delete[] saveOut;
	if (typeid(sFloat) == typeid(short4)) {
	  hipMemcpy(outNorm, saveOutNorm, norm_bytes, hipMemcpyHostToDevice);
	  delete[] saveOutNorm;
	}
      }
    }

    long long flops() const { return (x ? 1368ll : 1320ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };


  void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		     const int dagger, const QudaTwistGamma5Type twist) {
    if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
      a = 2.0 * kappa * mu;
      b = 1.0;
    } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
      a = -2.0 * kappa * mu;
      b = 1.0 / (1.0 + a*a);
    } else {
      errorQuda("Twist type %d not defined\n", twist);
    }
    if (dagger) a *= -1.0;

  }

  template <typename sFloat, typename gFloat>
  class TwistedDslashCuda : public SharedDslashCuda {

  private:
    const size_t bytes, norm_bytes;
    sFloat *out;
    float *outNorm;
    char *saveOut, *saveOutNorm;
    const sFloat *in, *x;
    const float *inNorm, *xNorm;
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const int dagger;
    double a;
    double b;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200)
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
#else
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }

  public:
    TwistedDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		      const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		      const sFloat *x, const float *xNorm, const double kappa, const double mu,
		      const double k, const int dagger, const size_t bytes, const size_t norm_bytes)
      : SharedDslashCuda(), bytes(bytes), norm_bytes(norm_bytes), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), in(in),
	inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm)
    { 
      bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
      setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);
      if (x) b *= k;
    }
    virtual ~TwistedDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      DSLASH(twistedMassDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     out, outNorm, gauge0, gauge1, in, inNorm, a, b, x, xNorm);
    }

    void preTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	saveOut = new char[bytes];
	hipMemcpy(saveOut, out, bytes, hipMemcpyDeviceToHost);
	if (typeid(sFloat) == typeid(short4)) {
	  saveOutNorm = new char[norm_bytes];
	  hipMemcpy(saveOutNorm, outNorm, norm_bytes, hipMemcpyDeviceToHost);
	}
      }
    }

    void postTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	hipMemcpy(out, saveOut, bytes, hipMemcpyHostToDevice);
	delete[] saveOut;
	if (typeid(sFloat) == typeid(short4)) {
	  hipMemcpy(outNorm, saveOutNorm, norm_bytes, hipMemcpyHostToDevice);
	  delete[] saveOutNorm;
	}
      }
    }

    long long flops() const { return (x ? 1416ll : 1392ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };


#ifdef DSLASH_PROFILING

#define TDIFF(a,b) 1e3*(b.tv_sec - a.tv_sec + 1e-6*(b.tv_usec - a.tv_usec))

  void dslashTimeProfile() {

    hipEventSynchronize(dslashEnd);
    float runTime;
    hipEventElapsedTime(&runTime, dslashStart, dslashEnd);
    dslashTime += runTime;

    for (int i=4; i>=0; i--) {
      if (!dslashParam.commDim[i] && i<4) continue;

      // kernel timing
      hipEventElapsedTime(&runTime, dslashStart, kernelStart[2*i]);
      kernelTime[2*i][0] += runTime; // start time
      hipEventElapsedTime(&runTime, dslashStart, kernelEnd[2*i]);
      kernelTime[2*i][1] += runTime; // end time
    }
      
#ifdef MULTI_GPU
    for (int i=3; i>=0; i--) {
      if (!dslashParam.commDim[i]) continue;

      for (int dir = 0; dir < 2; dir ++) {
	// pack timing
	hipEventElapsedTime(&runTime, dslashStart, packStart[2*i+dir]);
	packTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, packEnd[2*i+dir]);
	packTime[2*i+dir][1] += runTime; // end time
  
	// gather timing
	hipEventElapsedTime(&runTime, dslashStart, gatherStart[2*i+dir]);
	gatherTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, gatherEnd[2*i+dir]);
	gatherTime[2*i+dir][1] += runTime; // end time
      
	// comms timing
	runTime = TDIFF(dslashStart_h, commsStart[2*i+dir]);
	commsTime[2*i+dir][0] += runTime; // start time
	runTime = TDIFF(dslashStart_h, commsEnd[2*i+dir]);
	commsTime[2*i+dir][1] += runTime; // end time

	// scatter timing
	hipEventElapsedTime(&runTime, dslashStart, scatterStart[2*i+dir]);
	scatterTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, scatterEnd[2*i+dir]);
	scatterTime[2*i+dir][1] += runTime; // end time
      }
    }
#endif

  }

  void printDslashProfile() {
  
    printfQuda("Total Dslash time = %6.2f\n", dslashTime);

    char dimstr[8][8] = {"X-", "X+", "Y-", "Y+", "Z-", "Z+", "T-", "T+"};

    printfQuda("     %13s %13s %13s %13s %13s\n", "Pack", "Gather", "Comms", "Scatter", "Kernel");
    printfQuda("         %6s %6s %6s %6s %6s %6s %6s %6s %6s %6s\n", 
	       "Start", "End", "Start", "End", "Start", "End", "Start", "End", "Start", "End");

    printfQuda("%8s %55s %6.2f %6.2f\n", "Interior", "", kernelTime[8][0], kernelTime[8][1]);
      
    for (int i=3; i>=0; i--) {
      if (!dslashParam.commDim[i]) continue;

      for (int dir = 0; dir < 2; dir ++) {
	printfQuda("%8s ", dimstr[2*i+dir]);
#ifdef MULTI_GPU
	printfQuda("%6.2f %6.2f ", packTime[2*i+dir][0], packTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", gatherTime[2*i+dir][0], gatherTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", commsTime[2*i+dir][0], commsTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", scatterTime[2*i+dir][0], scatterTime[2*i+dir][1]);
#endif

	if (dir==0) printfQuda("%6.2f %6.2f\n", kernelTime[2*i][0], kernelTime[2*i][1]);
	else printfQuda("\n");
      }
    }

  }
#endif

  int gatherCompleted[Nstream];
  int previousDir[Nstream];
  int commsCompleted[Nstream];
  int commDimTotal;

  /**
   * Initialize the arrays used for the dynamic scheduling.
   */
  void initDslashCommsPattern() {
    for (int i=0; i<Nstream-1; i++) {
      gatherCompleted[i] = 0;
      commsCompleted[i] = 0;
    }
    gatherCompleted[Nstream-1] = 1;
    commsCompleted[Nstream-1] = 1;

    //   We need to know which was the previous direction in which
    //   communication was issued, since we only query a given event /
    //   comms call after the previous the one has successfully
    //   completed.
    for (int i=3; i>=0; i--) {
      if (dslashParam.commDim[i]) {
	int prev = Nstream-1;
	for (int j=3; j>i; j--) if (dslashParam.commDim[j]) prev = 2*j;
	previousDir[2*i + 1] = prev;
	previousDir[2*i + 0] = 2*i + 1; // always valid
      }
    }

    // this tells us how many events / comms occurances there are in
    // total.  Used for exiting the while loop
    commDimTotal = 0;
    for (int i=3; i>=0; i--) commDimTotal += dslashParam.commDim[i];
    commDimTotal *= 4; // 2 from pipe length, 2 from direction
  }

  void dslashCuda(DslashCuda &dslash, const size_t regSize, const int parity, const int dagger, 
		  const int volume, const int *faceVolumeCB) {

    dslashParam.parity = parity;
    dslashParam.kernel_type = INTERIOR_KERNEL;
    dslashParam.threads = volume;

    CUDA_EVENT_RECORD(dslashStart, 0);
    gettimeofday(&dslashStart_h, NULL);

#ifdef MULTI_GPU
    for(int i = 3; i >=0; i--){
      if (!dslashParam.commDim[i]) continue;

      // Record the start of the packing
      CUDA_EVENT_RECORD(packStart[2*i+0], streams[Nstream-1]);
      CUDA_EVENT_RECORD(packStart[2*i+1], streams[Nstream-1]);

      // Initialize pack from source spinor
      face->pack(*inSpinor, 1-parity, dagger, i, streams);
    
      // Record the end of the packing
      hipEventRecord(packEnd[2*i+0], streams[Nstream-1]);
      hipEventRecord(packEnd[2*i+1], streams[Nstream-1]);
    }

    for(int i = 3; i >=0; i--){
      if (!dslashParam.commDim[i]) continue;

      for (int dir=1; dir>=0; dir--) {
	hipStreamWaitEvent(streams[2*i+dir], packEnd[2*i+dir], 0);

	// Record the start of the gathering
	CUDA_EVENT_RECORD(gatherStart[2*i+dir], streams[2*i+dir]);

	// Initialize host transfer from source spinor
	face->gather(*inSpinor, dagger, 2*i+dir);

	// Record the end of the gathering
	hipEventRecord(gatherEnd[2*i+dir], streams[2*i+dir]);
      }
    }
#endif

    CUDA_EVENT_RECORD(kernelStart[Nstream-1], streams[Nstream-1]);
    dslash.apply(streams[Nstream-1]);
    CUDA_EVENT_RECORD(kernelEnd[Nstream-1], streams[Nstream-1]);

#ifdef MULTI_GPU
    initDslashCommsPattern();

    int completeSum = 0;
    while (completeSum < commDimTotal) {
      for (int i=3; i>=0; i--) {
	if (!dslashParam.commDim[i]) continue;
      
	for (int dir=1; dir>=0; dir--) {
	
	  // Query if gather has completed
	  if (!gatherCompleted[2*i+dir] && gatherCompleted[previousDir[2*i+dir]]) { 
	    if (hipSuccess == hipEventQuery(gatherEnd[2*i+dir])) {
	      gatherCompleted[2*i+dir] = 1;
	      completeSum++;
	      gettimeofday(&commsStart[2*i+dir], NULL);
	      face->commsStart(2*i+dir);
	    }
	  }
	
	  // Query if comms has finished
	  if (!commsCompleted[2*i+dir] && commsCompleted[previousDir[2*i+dir]] &&
	      gatherCompleted[2*i+dir]) {
	    if (face->commsQuery(2*i+dir)) { 
	      commsCompleted[2*i+dir] = 1;
	      completeSum++;
	      gettimeofday(&commsEnd[2*i+dir], NULL);
	    
	      // Record the end of the scattering
	      CUDA_EVENT_RECORD(scatterStart[2*i+dir], streams[2*i+dir]);
	    
	      // Scatter into the end zone
	      face->scatter(*inSpinor, dagger, 2*i+dir);
	    
	      // Record the end of the scattering
	      hipEventRecord(scatterEnd[2*i+dir], streams[2*i+dir]);
	    }
	  }

	}
      }
    
    }

    for (int i=3; i>=0; i--) {
      if (!dslashParam.commDim[i]) continue;

      dslashParam.kernel_type = static_cast<KernelType>(i);
      dslashParam.threads = dslash.Nface()*faceVolumeCB[i]; // updating 2 or 6 faces

      // wait for scattering to finish and then launch dslash
      hipStreamWaitEvent(streams[Nstream-1], scatterEnd[2*i], 0);
      hipStreamWaitEvent(streams[Nstream-1], scatterEnd[2*i+1], 0);

      CUDA_EVENT_RECORD(kernelStart[2*i], streams[Nstream-1]);
      dslash.apply(streams[Nstream-1]); // all faces use this stream
      CUDA_EVENT_RECORD(kernelEnd[2*i], streams[Nstream-1]);
    }

    CUDA_EVENT_RECORD(dslashEnd, 0);
    DSLASH_TIME_PROFILE();

#endif // MULTI_GPU
  }

  // Wilson wrappers
  void wilsonDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const cudaColorSpinorField *in, const int parity,
			const int dagger, const cudaColorSpinorField *x, const double &k, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge %d and spinor %d precision not supported", 
		gauge.Precision(), in->Precision());

    const void *xv = (x ? x->V() : 0);
    const void *xn = (x ? x->Norm() : 0);

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new WilsonDslashCuda<double2, double2>((double2*)out->V(), (float*)out->Norm(), 
						      (double2*)gauge0, (double2*)gauge1, 
						      gauge.Reconstruct(), (double2*)in->V(), 
						      (float*)in->Norm(), (double2*)xv, (float*)xn,
						      k, dagger, in->Bytes(), in->NormBytes());
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new WilsonDslashCuda<float4, float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, (float4*)gauge1,
						    gauge.Reconstruct(), (float4*)in->V(), (float*)in->Norm(), 
						    (float4*)xv, (float*)xn, k, dagger, in->Bytes(), in->NormBytes());
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new WilsonDslashCuda<short4, short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, (short4*)gauge1,
						    gauge.Reconstruct(), (short4*)in->V(), (float*)in->Norm(),
						    (short4*)xv, (float*)xn, k, dagger, in->Bytes(), in->NormBytes());
    }
    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

  }



  void twistedMassDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			     const cudaColorSpinorField *in, const int parity, const int dagger, 
			     const cudaColorSpinorField *x, const double &kappa, const double &mu, 
			     const double &a, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_TWISTED_MASS_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    const void *xv = x ? x->V() : 0;
    const void *xn = x ? x->Norm() : 0;

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new TwistedDslashCuda<double2,double2>((double2*)out->V(), (float*)out->Norm(), (double2*)gauge0, 
						      (double2*)gauge1, gauge.Reconstruct(), (double2*)in->V(), 
						      (float*)in->Norm(), (double2*)xv, (float*)xn, 
						      kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new TwistedDslashCuda<float4,float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, (float4*)gauge1, 
						    gauge.Reconstruct(), (float4*)in->V(), (float*)in->Norm(), 
						    (float4*)xv, (float*)xn, kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new TwistedDslashCuda<short4,short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, (short4*)gauge1, 
						    gauge.Reconstruct(), (short4*)in->V(), (float*)in->Norm(), 
						    (short4*)xv, (float*)xn, kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
    
    }

    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Twisted mass dslash has not been built");
#endif

  }


  template <typename sFloat>
  class TwistGamma5Cuda : public Tunable {

  private:
    sFloat *out;
    float *outNorm;
    sFloat *in;
    float *inNorm;
    double a;
    double b;
    size_t bytes;
    size_t norm_bytes;

    int sharedBytesPerThread() const { return 0; }
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool advanceGridDim(TuneParam &param) const { return false; } // Don't tune the grid dimensions.

    char *saveOut, *saveOutNorm;

  public:
    TwistGamma5Cuda(sFloat *out, float *outNorm, sFloat *in, float *inNorm,
		    double kappa, double mu, const int dagger, 
		    QudaTwistGamma5Type twist, size_t bytes, size_t norm_bytes) :
      out(out), outNorm(outNorm), in(in), inNorm(inNorm), 
      bytes(bytes), norm_bytes(norm_bytes){
      bindSpinorTex(bytes, norm_bytes, in, inNorm);
      setTwistParam(a, b, kappa, mu, dagger, twist);
    }
    virtual ~TwistGamma5Cuda() {
      unbindSpinorTex(in, inNorm);    
    }

    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << dslashConstants.x[0] << "x";
      vol << dslashConstants.x[1] << "x";
      vol << dslashConstants.x[2] << "x";
      vol << dslashConstants.x[3];    
      return TuneKey(vol.str(), typeid(*this).name(), aux.str());
    }  

    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      twistGamma5Kernel<<<gridDim, tp.block, tp.shared_bytes, stream>>> 
	(out, outNorm, a, b, in, inNorm, dslashParam);
    }

    void preTune() {
      saveOut = new char[bytes];
      hipMemcpy(saveOut, out, bytes, hipMemcpyDeviceToHost);
      if (typeid(sFloat) == typeid(short4)) {
	saveOutNorm = new char[norm_bytes];
	hipMemcpy(saveOutNorm, outNorm, norm_bytes, hipMemcpyDeviceToHost);
      }
    }

    void postTune() {
      hipMemcpy(out, saveOut, bytes, hipMemcpyHostToDevice);
      delete[] saveOut;
      if (typeid(sFloat) == typeid(short4)) {
	hipMemcpy(outNorm, saveOutNorm, norm_bytes, hipMemcpyHostToDevice);
	delete[] saveOutNorm;
      }
    }

    std::string paramString(const TuneParam &param) const {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 24ll * dslashConstants.VolumeCB(); }
  };

  void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		       const int dagger, const double &kappa, const double &mu,
		       const QudaTwistGamma5Type twist)
  {
    dslashParam.threads = in->Volume();

#ifdef GPU_TWISTED_MASS_DIRAC
    Tunable *twistGamma5 = 0;

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      twistGamma5 = new TwistGamma5Cuda<double2>
	((double2*)out->V(), (float*)out->Norm(), (double2*)in->V(), 
	 (float*)in->Norm(), kappa, mu, dagger, twist, in->Bytes(), in->NormBytes());
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      twistGamma5 = new TwistGamma5Cuda<float4>
	((float4*)out->V(), (float*)out->Norm(), (float4*)in->V(), 
	 (float*)in->Norm(), kappa, mu, dagger, twist, in->Bytes(), in->NormBytes());
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      twistGamma5 = new TwistGamma5Cuda<short4>
	((short4*)out->V(), (float*)out->Norm(), (short4*)in->V(), 
	 (float*)in->Norm(), kappa, mu, dagger, twist, in->Bytes(), in->NormBytes());
    }

    twistGamma5->apply(streams[Nstream-1]);
    checkCudaError();

    delete twistGamma5;
#else
    errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

} // namespace quda

#include "misc_helpers.cu"


