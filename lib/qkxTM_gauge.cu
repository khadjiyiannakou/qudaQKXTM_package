#include "hip/hip_runtime.h"
#include <qkxTM.h>
#include <mpi.h>
#include <cuPrintf.cu>

#define THREADS_PER_BLOCK 32

using namespace quda;


/* texture block */

texture<int4, 1> gaugeTexPlaq;      // use this texture reference only for plaquette
texture<int4, 1> gaugeTexAPE;    // use this for APE smearing , this texture will be binded and unbinded   
texture<int4, 1> vectorTex;
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/* block for device constants */
__constant__ bool c_dimBreak[4];
__constant__ int c_nColor;
__constant__ int c_nDim;
__constant__ int c_localL[4];
__constant__ int c_plusGhost[4];
__constant__ int c_minusGhost[4];
__constant__ int c_stride;
__constant__ int c_surface[4];

__constant__ int c_nSpin;
__constant__ double c_alphaAPE;
__constant__ double c_alphaGauss;
__constant__ int c_threads;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

static DeviceConstants dev_constants;          // static global structure to take care of device constants
static bool device_constants_flag = false;     // to check if we already cpy constants to device 


                                                           /* Area for kernels */

#if (__COMPUTE_CAPABILITY__ >= 130)
__inline__ __device__ double2 fetch_double2(texture<int4, 1> t, int i)
{
  int4 v = tex1Dfetch(t,i);
  return make_double2(__hiloint2double(v.y, v.x), __hiloint2double(v.w, v.z));
}
#else
__inline__ __device__ double2 fetch_double2(texture<int4, 1> t, int i)
{
  // do nothing 
  return make_double2(0.0, 0.0);
}
#endif


__global__ void calculatePlaq_kernel(double *partial_plaq){

#include <plaquette_core.h>

}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void quda::device_constants(SmearingInfo *smearInfo){
  hipError_t error;

  int nDim = 4;
  int localL[QUDA_MAX_DIM];
  int totalL[QUDA_MAX_DIM];
  int localVolume;              // whole volume not even odd
  int totalVolume;
  int surface[QUDA_MAX_DIM];
  int strideFull;          // stride for the whole volume
  int nProc[QUDA_MAX_DIM];
  int plusGhost[QUDA_MAX_DIM];
  int minusGhost[QUDA_MAX_DIM];
  bool dimBreak[QUDA_MAX_DIM];

  if(device_constants_flag == false){

    for(int i = 0 ; i < nDim ; i++)
      nProc[i] = comm_dim(i);        // save number of procs in each direction

    for(int i = 0 ; i < nDim ; i++){   // take local and total lattice
      localL[i] = smearInfo->lL[i];
      totalL[i] = nProc[i] * localL[i];
    }
  
    localVolume = 1;
    totalVolume = 1;
    for(int i = 0 ; i < nDim ; i++){
      localVolume *= localL[i];
      totalVolume *= totalL[i];
    }

    strideFull = localVolume;

    for (int i=0; i<nDim; i++) {
      surface[i] = 1;
      for (int j=0; j<nDim; j++) {
	if (i==j) continue;
	surface[i] *= localL[j];
      }
    }
    
    for(int i = 0 ; i < nDim ; i++){
      plusGhost[i] =0;
      minusGhost[i] = 0;
    }
    
#ifdef MULTI_GPU
    int lastIndex = localVolume;
    for(int i = 0 ; i < nDim ; i++)
      if( localL[i] < totalL[i] ){
	plusGhost[i] = lastIndex ;
	minusGhost[i] = lastIndex + surface[i];
	lastIndex += 2*surface[i];
      }
#endif
    

    for(int i = 0 ; i < nDim ; i++){
      if( localL[i] < totalL[i])
	dimBreak[i] = true;
      else
	dimBreak[i] = false;
    }
  
    
    dev_constants.nColor = 3;
    dev_constants.nSpin = 4;
    dev_constants.nDim = nDim;
    dev_constants.threads = localVolume;
    dev_constants.alphaAPE = smearInfo->alphaAPE;
    dev_constants.alphaGauss = smearInfo->alphaGauss;
    for(int i = 0 ; i < nDim ; i++){
      dev_constants.localL[i] = localL[i];
      dev_constants.plusGhost[i] = plusGhost[i];
      dev_constants.minusGhost[i] = minusGhost[i];
      dev_constants.dimBreak[i] = dimBreak[i];
    }

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_dimBreak),dev_constants.dimBreak, 4*sizeof(bool) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_nColor),&(dev_constants.nColor), sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_nSpin),&(dev_constants.nSpin), sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_threads),&(dev_constants.threads), sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_alphaAPE),&(dev_constants.alphaAPE), sizeof(double) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_alphaGauss),&(dev_constants.alphaGauss), sizeof(double) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_nDim),&(dev_constants.nDim), sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_plusGhost),dev_constants.plusGhost, 4*sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_minusGhost),dev_constants.minusGhost, 4*sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_localL),dev_constants.localL, 4*sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_stride),&strideFull, sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    error = hipMemcpyToSymbol(HIP_SYMBOL(c_surface),surface, 4*sizeof(int) );
    if(error != hipSuccess) errorQuda("Problem copy to symbol in device memory");

    device_constants_flag = true;
  }

}



/////////////////////////////////////////////////////////////////////////////// for gauge field /////////////////////////////////////////
QKXTM_Gauge::QKXTM_Gauge(SmearingInfo *smearInfo):
  nColor(3) , nSpin(4) , nDim(4) , nsmearAPE(smearInfo->nsmearAPE),
  nsmearGauss(smearInfo->nsmearGauss), alphaAPE(smearInfo->alphaAPE),
  alphaGauss( smearInfo->alphaGauss) , init(false) , h_elem(NULL) , d_elem(NULL),
  init_host_alloc(false) , init_device_alloc(false) , init_ext_host_ghost(false) , 
  init_ext_device_ghost(false) , h_ext_ghost(NULL), d_ext_ghost(NULL),
  gauge_binded_plaq(false), packGauge_flag(false), loadGauge_flag(false),
   gauge_binded_ape(false)
{



  for(int i = 0 ; i < nDim ; i++)
    nProc[i] = comm_dim(i);        // save number of procs in each direction

  for(int i = 0 ; i < nDim ; i++){   // take local and total lattice
    localL[i] = smearInfo->lL[i];
    totalL[i] = nProc[i] * localL[i];
  }
  
  localVolume = 1;
  totalVolume = 1;
  for(int i = 0 ; i < nDim ; i++){
    localVolume *= localL[i];
    totalVolume *= totalL[i];
  }

  strideFull = localVolume;

  for (int i=0; i<nDim; i++) {
    surface[i] = 1;
    for (int j=0; j<nDim; j++) {
      if (i==j) continue;
      surface[i] *= localL[j];
    }
  }

  for(int i = 0 ; i < nDim ; i++)
    if( localL[i] == totalL[i] )
      surface[i] = 0;
  
  ghost_length = 0;

  for(int i = 0 ; i < nDim ; i++)
    ghost_length += 2*surface[i];
  
  total_length = localVolume + ghost_length;
  
  bytes_total_length = total_length*nColor*nColor*nDim*2*sizeof(double);
  bytes_ghost_length = ghost_length*nColor*nColor*nDim*2*sizeof(double);

  for(int i = 0 ; i < nDim ; i++){
    plusGhost[i] =0;
    minusGhost[i] = 0;
  }
  
#ifdef MULTI_GPU
  int lastIndex = localVolume;
  for(int i = 0 ; i < nDim ; i++)
    if( localL[i] < totalL[i] ){
      plusGhost[i] = lastIndex ;
      minusGhost[i] = lastIndex + surface[i];
      lastIndex += 2*surface[i];
    }
#endif

  init = true;
}

QKXTM_Gauge::~QKXTM_Gauge(){
  destroy_all();
  unbindGaugePlaq();
  packGauge_flag = false;
  loadGauge_flag = false;
}

void QKXTM_Gauge::create_host(){

  if(init_host_alloc == false){
    h_elem = (double*) malloc(bytes_total_length);
    if(h_elem == NULL) errorQuda("Error with allocation host memory");
  }
  init_host_alloc = true;
}

void QKXTM_Gauge::create_host_ghost(){
#ifdef MULTI_GPU
  if(init_ext_host_ghost == false){
    if( comm_size() > 1){
      h_ext_ghost = (double*) malloc(bytes_ghost_length);
      if(h_ext_ghost == NULL)errorQuda("Error with allocation host memory");
    }
  }
#endif
  init_ext_host_ghost = true;
}


void QKXTM_Gauge::create_device(){
  hipError_t error;
  if(init_device_alloc == false){
    error = hipMalloc((void**)&d_elem,bytes_total_length);
    if( error != hipSuccess) errorQuda("Error with allocation device memory");
  }
  init_device_alloc = true;
}

void QKXTM_Gauge::create_device_ghost(){
#ifdef MULTI_GPU
  hipError_t error;
  if(init_ext_device_ghost == false){
    if( comm_size() > 1){
      error = hipMalloc((void**)&d_ext_ghost,bytes_ghost_length);
      if( error != hipSuccess) errorQuda("Error with allocation device memory");
    }
  }
#endif
  init_ext_device_ghost = true;
}

void QKXTM_Gauge::destroy_host(){
  if(init_host_alloc == true){
    free(h_elem);
    h_elem = NULL;
    init_host_alloc = false;
  }
}

void QKXTM_Gauge::destroy_device(){
  if(init_device_alloc == true){
    hipFree(d_elem);
    d_elem = NULL;
    init_device_alloc = false;
  }
}

void QKXTM_Gauge::destroy_host_ghost(){
#ifdef MULTI_GPU
  if( (comm_size() > 1) && (init_ext_host_ghost == true) ){
    free(h_ext_ghost);
  } 
#endif
  init_ext_host_ghost = false;
}

void QKXTM_Gauge::destroy_device_ghost(){
#ifdef MULTI_GPU
  if( (comm_size() > 1) && (init_ext_device_ghost == true) ){
    hipFree(d_ext_ghost);
  }
#endif
  init_ext_device_ghost = false;
}

void QKXTM_Gauge::create_all(){
  create_host();
  create_host_ghost();
  create_device();
  create_device_ghost();
}

void QKXTM_Gauge::destroy_all(){
  destroy_host();
  destroy_host_ghost();
  destroy_device();
  destroy_device_ghost();
}

void QKXTM_Gauge::packGauge(void **gauge){

  if(init_host_alloc == true && packGauge_flag == false){
    double **p_gauge = (double**) gauge;
    
    for(int dir = 0 ; dir < nDim ; dir++)
      for(int iv = 0 ; iv < localVolume ; iv++)
	for(int c1 = 0 ; c1 < nColor ; c1++)
	  for(int c2 = 0 ; c2 < nColor ; c2++)
	    for(int part = 0 ; part < 2 ; part++){
	      h_elem[dir*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + iv*2 + part] = p_gauge[dir][iv*nColor*nColor*2 + c1*nColor*2 + c2*2 + part];
	    }

    printfQuda("Gauge qkxTM packed on gpu form\n");
    packGauge_flag = true;
  }
  else{
    errorQuda("Error not create host pointer");
  }

}

void QKXTM_Gauge::loadGauge(){

  if(packGauge_flag == true && loadGauge_flag == false){
    hipError_t error;
    if( (init_host_alloc == true) && (init_device_alloc == true) ){
      error = hipMemcpy(d_elem,h_elem,(bytes_total_length - bytes_ghost_length), hipMemcpyHostToDevice );
      if(error != hipSuccess) errorQuda("Error problem with load Gauge on device");
      loadGauge_flag = true;
      printfQuda("Gauge qkxTM loaded on gpu\n");
    }
    else{
      errorQuda("Error try to load gauge without allocate properly first");
    }
  }

}

void QKXTM_Gauge::ghostToHost(){   // gpu collect ghost and send it to host

  // direction x ////////////////////////////////////
#ifdef MULTI_GPU
  if( localL[0] < totalL[0]){
    int position;
    int height = localL[1] * localL[2] * localL[3]; // number of blocks that we need
    size_t width = 2*sizeof(double);
    size_t spitch = localL[0]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;
  // set plus points to minus area
    position = (localL[0]-1)*localL[1]*localL[2]*localL[3];
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[0]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[0]*2 + c1*nColor*surface[0]*2 + c2*surface[0]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}
  // set minus points to plus area
    position = 0;
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[0]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[0]*2 + c1*nColor*surface[0]*2 + c2*surface[0]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}


  }
  // direction y ///////////////////////////////////

  if( localL[1] < totalL[1]){

    int position;
    int height = localL[2] * localL[3]; // number of blocks that we need
    size_t width = localL[0]*2*sizeof(double);
    size_t spitch = localL[1]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;

  // set plus points to minus area
    position = localL[0]*(localL[1]-1)*localL[2]*localL[3];
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[1]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[1]*2 + c1*nColor*surface[1]*2 + c2*surface[1]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}
  // set minus points to plus area
    position = 0;
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[1]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[1]*2 + c1*nColor*surface[1]*2 + c2*surface[1]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}


  }
  
  // direction z //////////////////////////////////
  if( localL[2] < totalL[2]){

    int position;
    int height = localL[3]; // number of blocks that we need
    size_t width = localL[1]*localL[0]*2*sizeof(double);
    size_t spitch = localL[2]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;

  // set plus points to minus area
    position = localL[0]*localL[1]*(localL[2]-1)*localL[3];
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[2]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[2]*2 + c1*nColor*surface[2]*2 + c2*surface[2]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}
  // set minus points to plus area
    position = 0;
    for(int i = 0 ; i < nDim ; i++)
      for(int c1 = 0 ; c1 < nColor ; c1++)
	for(int c2 = 0 ; c2 < nColor ; c2++){
	  d_elem_offset = d_elem + i*nColor*nColor*localVolume*2 + c1*nColor*localVolume*2 + c2*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[2]*nDim*nColor*nColor*2 + i*nColor*nColor*surface[2]*2 + c1*nColor*surface[2]*2 + c2*surface[2]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}

  }

  //  printfQuda("before copy device to host\n");  
  // direction t /////////////////////////////////////
  if( localL[3] < totalL[3]){
    int position;
    int height = nDim*nColor*nColor;
    size_t width = localL[2]*localL[1]*localL[0]*2*sizeof(double);
    size_t spitch = localL[3]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;
  // set plus points to minus area
    position = localL[0]*localL[1]*localL[2]*(localL[3]-1);
    d_elem_offset = d_elem + position*2;
    h_elem_offset = h_elem + minusGhost[3]*nDim*nColor*nColor*2;
    hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
  // set minus points to plus area
    position = 0;
    d_elem_offset = d_elem + position*2;
    h_elem_offset = h_elem + plusGhost[3]*nDim*nColor*nColor*2;
    hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
  }


#endif
}

void QKXTM_Gauge::cpuExchangeGhost(){ // cpus exchange links

#ifdef MULTI_GPU
  if(comm_size() > 1){

    MPI_Request request_recv[2*nDim];
    MPI_Request request_send[2*nDim];
    int back_nbr[4] = {X_BACK_NBR,Y_BACK_NBR,Z_BACK_NBR,T_BACK_NBR};             
    int fwd_nbr[4] = {X_FWD_NBR,Y_FWD_NBR,Z_FWD_NBR,T_FWD_NBR};

    // direction x
    if(localL[0] < totalL[0]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[0]*nColor*nColor*nDim*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[0]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + minusGhost[0]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[0], 0, &(request_recv[0]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[0], 0, &(request_send[0]));
      comm_wait(&(request_recv[0])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[0]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + plusGhost[0]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[0], 1, &(request_recv[1]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[0], 1, &(request_send[1]));
      comm_wait(&(request_recv[1])); // blocking until receive finish
      
    }
    // direction y
    if(localL[1] < totalL[1]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[1]*nColor*nColor*nDim*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[1]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + minusGhost[1]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[1], 2, &(request_recv[2]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[1], 2, &(request_send[2]));
      comm_wait(&(request_recv[2])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[1]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + plusGhost[1]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[1], 3, &(request_recv[3]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[1], 3, &(request_send[3]));
      comm_wait(&(request_recv[3])); // blocking until receive finish
    }

    // direction z
    if(localL[2] < totalL[2]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[2]*nColor*nColor*nDim*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[2]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + minusGhost[2]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[2], 4, &(request_recv[4]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[2], 4, &(request_send[4]));
      comm_wait(&(request_recv[4])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[2]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + plusGhost[2]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[2], 5, &(request_recv[5]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[2], 5, &(request_send[5]));
      comm_wait(&(request_recv[5])); // blocking until receive finish
    }


    // direction t
    if(localL[3] < totalL[3]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[3]*nColor*nColor*nDim*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[3]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + minusGhost[3]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[3], 6, &(request_recv[6]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[3], 6, &(request_send[6]));
      comm_wait(&(request_recv[6])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[3]-localVolume)*nColor*nColor*nDim*2;
      pointer_send = h_elem + plusGhost[3]*nColor*nColor*nDim*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[3], 7, &(request_recv[7]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[3], 7, &(request_send[7]));
      comm_wait(&(request_recv[7])); // blocking until receive finish
    }


  }
#endif

}

void QKXTM_Gauge::ghostToDevice(){ // simple cudamemcpy to send ghost to device
#ifdef MULTI_GPU
  if(comm_size() > 1){
    double *host = h_ext_ghost;
    double *device = d_elem + localVolume*nColor*nColor*nDim*2;
    hipMemcpy(device,host,bytes_ghost_length,hipMemcpyHostToDevice);
  }
#endif
}

void QKXTM_Gauge::bindGaugePlaq(){

  hipError_t error;
  if( gauge_binded_plaq == false ){
    error = hipBindTexture(0,gaugeTexPlaq,d_elem,bytes_total_length);
    if(error != hipSuccess) errorQuda("Problem bind Texture");
  }
  
  gauge_binded_plaq = true;
}

void QKXTM_Gauge::unbindGaugePlaq(){

  hipError_t error;
  if(gauge_binded_plaq == true){
        error = hipUnbindTexture(gaugeTexPlaq);
    if(error != hipSuccess) errorQuda("Problem unbind Texture");
  }

  gauge_binded_plaq = false;
}



double QKXTM_Gauge::calculatePlaq(void **gauge){


  create_all();
  //  if(device_constants_flag == false) device_constants(paramKernel);
  if(device_constants_flag == false) errorQuda("Forgot copy device constants");
  if(gauge_binded_plaq == false) bindGaugePlaq();
  if(packGauge_flag == false) packGauge(gauge);
  if(loadGauge_flag == false) loadGauge();

  ghostToHost(); // collect surface from device and send it to host
  cpuExchangeGhost(); // cpus exchange surfaces with previous and forward proc all dir
  ghostToDevice();   // now the host send surface to device 

  dim3 blockDim( THREADS_PER_BLOCK , 1, 1);
  dim3 gridDim( (localVolume + blockDim.x -1)/blockDim.x , 1 , 1);

  double *h_partial_plaq = NULL;
  double *d_partial_plaq = NULL;

  h_partial_plaq = (double*) malloc(gridDim.x * sizeof(double) ); // only real part
  if(h_partial_plaq == NULL) errorQuda("Error allocate memory for host partial plaq");

  hipError_t error;
  error = hipMalloc((void**)&d_partial_plaq, gridDim.x * sizeof(double));
  if(error != hipSuccess) errorQuda("Error allocate device memory for partial plaq");

  // cudaPrintfInit();

  hipEvent_t start,stop;
  float elapsedTime;
  error = hipEventCreate(&start);
  if(error != hipSuccess) errorQuda("Problem create Event for plaquette kernel");
  error = hipEventCreate(&stop);
  if(error != hipSuccess) errorQuda("Problem create Event for plaquette kernel");
  
  error = hipEventRecord(start,0);
  if(error != hipSuccess) errorQuda("Problem record Event for plaquette kernel");


    calculatePlaq_kernel<<<gridDim,blockDim>>>(d_partial_plaq);

  if ( hipSuccess != hipGetLastError() ) errorQuda("Problem executing plaquette kernel");

  error = hipEventRecord(stop,0);
  if(error != hipSuccess) errorQuda("Problem record Event for plaquette kernel");

  error = hipEventSynchronize(stop);
  if(error != hipSuccess) errorQuda("Problem synch Event for plaquette kernel");

  error = hipEventElapsedTime(&elapsedTime,start,stop);
  if(error != hipSuccess) errorQuda("Problem take timing for plaquette kernel");
  //  if(comm_rank() == 0)  cudaPrintfDisplay(stdout,true);
  //  cudaPrintfEnd();

  error = hipEventDestroy(start);
  if(error != hipSuccess) errorQuda("Problem destroy event for plaquette kernel");

  error = hipEventDestroy(stop);
  if(error != hipSuccess) errorQuda("Problem destroy event for plaquette kernel");

  printfQuda("Elapsed time for plaquette kernel is %f ms\n",elapsedTime);

  // now copy partial plaq to host
  error = hipMemcpy(h_partial_plaq, d_partial_plaq , gridDim.x * sizeof(double) , hipMemcpyDeviceToHost);
  if( error != hipSuccess ) errorQuda("Error copy partial plaq from device to host");

  double plaquette = 0.;

#ifdef MULTI_GPU
  double globalPlaquette = 0.;
#endif
  // simple host reduction on plaq
  for(int i = 0 ; i < gridDim.x ; i++)
    plaquette += h_partial_plaq[i];



  free(h_partial_plaq);
  hipFree(d_partial_plaq);

  unbindGaugePlaq();

#ifdef MULTI_GPU
  int rc = MPI_Allreduce(&plaquette , &globalPlaquette , 1 , MPI_DOUBLE , MPI_SUM , MPI_COMM_WORLD);
  if( rc != MPI_SUCCESS ) errorQuda("Error in MPI reduction for plaquette");
  return globalPlaquette/(totalVolume*nColor*6) ;
#else
  return plaquette/(totalVolume*nColor*6);
#endif  
  


}


void QKXTM_Gauge::bindGaugeAPE(){

  hipError_t error;

  if( gauge_binded_ape == false ){
    error = hipBindTexture(0,gaugeTexAPE,d_elem,bytes_total_length);
    if(error != hipSuccess) errorQuda("Problem bind Texture");
  }
  
  gauge_binded_ape = true;
}

void QKXTM_Gauge::unbindGaugeAPE(){

  hipError_t error;
  if(gauge_binded_ape == true){
        error = hipUnbindTexture(gaugeTexAPE);
    if(error != hipSuccess) errorQuda("Problem unbind Texture");
  }

  gauge_binded_ape = false;
}


void QKXTM_Gauge::rebindGaugeAPE(){

  hipError_t error;

  error = hipUnbindTexture(gaugeTexAPE);
  if(error != hipSuccess) errorQuda("Problem unbind Texture");
  error = hipBindTexture(0,gaugeTexAPE,d_elem,bytes_total_length);
  if(error != hipSuccess) errorQuda("Problem bind Texture");

}


void QKXTM_Gauge::printInfo(){
  printfQuda("Number of colors is %d\n",nColor);
  printfQuda("Number of spins is %d\n",nSpin);
  printfQuda("Number of dimensions is %d\n",nDim);
  printfQuda("Number of process in each direction is (x,y,z,t) %d x %d x %d x %d\n",nProc[0],nProc[1],nProc[2],nProc[3]);
  printfQuda("Total lattice is (x,y,z,t) %d x %d x %d x %d\n",totalL[0],totalL[1],totalL[2],totalL[3]);
  printfQuda("Local lattice is (x,y,z,t) %d x %d x %d x %d\n",localL[0],localL[1],localL[2],localL[3]);
  printfQuda("Total volume is %d\n",totalVolume);
  printfQuda("Local volume is %d\n",localVolume);
  printfQuda("Surface is (x,y,z,t) ( %d , %d , %d , %d)\n",surface[0],surface[1],surface[2],surface[3]);
  printfQuda("Stride for GPU use is %d\n",strideFull);
  printfQuda("Ghost length with out ndim*ncolor*ncolor*2 is %d\n",ghost_length);
  printfQuda("Total length with out ndim*ncolor*ncolor*2 is %d\n",total_length);
  printfQuda("GPU memory needed is %f MB \n",bytes_total_length/(1024.0 * 1024.0));
  printfQuda("The plus Ghost points in directions (x,y,z,t) ( %d , %d , %d , %d )\n",plusGhost[0],plusGhost[1],plusGhost[2],plusGhost[3]);
  printfQuda("The Minus Ghost points in directixons (x,y,z,t) ( %d , %d , %d , %d )\n",minusGhost[0],minusGhost[1],minusGhost[2],minusGhost[3]);
  printfQuda("For APE smearing we use nsmear = %d , alpha = %lf\n",nsmearAPE,alphaAPE);
  printfQuda("For Gauss smearing we use nsmear = %d , alpha = %lf\n",nsmearGauss,alphaGauss);

}

void quda::APE_smearing(QKXTM_Gauge &gaugeAPE , QKXTM_Gauge &gaugeTmp){// this is a function not a routine which perform smearing , need two QKXTM_Gauge objects

 
  if(device_constants_flag == false) errorQuda("Forgot copy device constants"); //check for device constant ok

  // TODO : need propagator class

  
}

/////////////////////////////////////////////////////////////////// for vector /////////////////////////////////////////

QKXTM_Vector::QKXTM_Vector(SmearingInfo *smearInfo):
  nColor(3) , nSpin(4) , nDim(4) , nsmearAPE(smearInfo->nsmearAPE),
  nsmearGauss(smearInfo->nsmearGauss), alphaAPE(smearInfo->alphaAPE),
  alphaGauss( smearInfo->alphaGauss) , init(false) , h_elem(NULL) , d_elem(NULL),
  init_host_alloc(false) , init_device_alloc(false) , init_ext_host_ghost(false) , 
  init_ext_device_ghost(false) , h_ext_ghost(NULL), d_ext_ghost(NULL),
  vector_binded(false), packVector_flag(false), loadVector_flag(false)
{
  
  for(int i = 0 ; i < nDim ; i++)
    nProc[i] = comm_dim(i);        // save number of procs in each direction

  for(int i = 0 ; i < nDim ; i++){   // take local and total lattice
    localL[i] = smearInfo->lL[i];
    totalL[i] = nProc[i] * localL[i];
  }
  
  localVolume = 1;
  totalVolume = 1;
  for(int i = 0 ; i < nDim ; i++){
    localVolume *= localL[i];
    totalVolume *= totalL[i];
  }

  strideFull = localVolume;

  for (int i=0; i<nDim; i++) {
    surface[i] = 1;
    for (int j=0; j<nDim; j++) {
      if (i==j) continue;
      surface[i] *= localL[j];
    }
  }

  for(int i = 0 ; i < nDim ; i++)
    if( localL[i] == totalL[i] )
      surface[i] = 0;
  
  ghost_length = 0;

  for(int i = 0 ; i < nDim ; i++)
    ghost_length += 2*surface[i];
  
  total_length = localVolume + ghost_length;
  
  bytes_total_length = total_length*nSpin*nColor*2*sizeof(double); // for vector is different from gauge
  bytes_ghost_length = ghost_length*nSpin*nColor*2*sizeof(double);

  for(int i = 0 ; i < nDim ; i++){
    plusGhost[i] =0;
    minusGhost[i] = 0;
  }
  
#ifdef MULTI_GPU
  int lastIndex = localVolume;
  for(int i = 0 ; i < nDim ; i++)
    if( localL[i] < totalL[i] ){
      plusGhost[i] = lastIndex ;
      minusGhost[i] = lastIndex + surface[i];
      lastIndex += 2*surface[i];
    }
#endif

  init = true;
}

QKXTM_Vector::~QKXTM_Vector(){
  destroy_all();
  unbindVector();
  packVector_flag = false;
  loadVector_flag = false;
}

void QKXTM_Vector::create_host(){

  if(init_host_alloc == false){
    h_elem = (double*) malloc(bytes_total_length);
    if(h_elem == NULL) errorQuda("Error with allocation host memory");
  }
  init_host_alloc = true;
}

void QKXTM_Vector::create_host_ghost(){
#ifdef MULTI_GPU
  if(init_ext_host_ghost == false){
    if( comm_size() > 1){
      h_ext_ghost = (double*) malloc(bytes_ghost_length);
      if(h_ext_ghost == NULL)errorQuda("Error with allocation host memory");
    }
  }
#endif
  init_ext_host_ghost = true;
}


void QKXTM_Vector::create_device(){
  hipError_t error;
  if(init_device_alloc == false){
    error = hipMalloc((void**)&d_elem,bytes_total_length);
    if( error != hipSuccess) errorQuda("Error with allocation device memory");
  }
  init_device_alloc = true;
}

void QKXTM_Vector::create_device_ghost(){
#ifdef MULTI_GPU
  hipError_t error;
  if(init_ext_device_ghost == false){
    if( comm_size() > 1){
      error = hipMalloc((void**)&d_ext_ghost,bytes_ghost_length);
      if( error != hipSuccess) errorQuda("Error with allocation device memory");
    }
  }
#endif
  init_ext_device_ghost = true;
}

void QKXTM_Vector::destroy_host(){
  if(init_host_alloc == true){
    free(h_elem);
    h_elem = NULL;
    init_host_alloc = false;
  }
}

void QKXTM_Vector::destroy_device(){
  if(init_device_alloc == true){
    hipFree(d_elem);
    d_elem = NULL;
    init_device_alloc = false;
  }
}

void QKXTM_Vector::destroy_host_ghost(){
#ifdef MULTI_GPU
  if( (comm_size() > 1) && (init_ext_host_ghost == true) ){
    free(h_ext_ghost);
  } 
#endif
  init_ext_host_ghost = false;
}

void QKXTM_Vector::destroy_device_ghost(){
#ifdef MULTI_GPU
  if( (comm_size() > 1) && (init_ext_device_ghost == true) ){
    hipFree(d_ext_ghost);
  }
#endif
  init_ext_device_ghost = false;
}

void QKXTM_Vector::create_all(){
  create_host();
  create_host_ghost();
  create_device();
  create_device_ghost();
}

void QKXTM_Vector::destroy_all(){
  destroy_host();
  destroy_host_ghost();
  destroy_device();
  destroy_device_ghost();
}

void QKXTM_Vector::packVector(void *vector){

  if(init_host_alloc == true && packVector_flag == false){
    double *p_vector = (double*) vector;
    
      for(int iv = 0 ; iv < localVolume ; iv++)
	for(int mu = 0 ; mu < nSpin ; mu++)                // always work with format colors inside spins
	  for(int c1 = 0 ; c1 < nColor ; c1++)
	    for(int part = 0 ; part < 2 ; part++){
	      h_elem[mu*nColor*localVolume*2 + c1*localVolume*2 + iv*2 + part] = p_vector[iv*nSpin*nColor*2 + mu*nColor*2 + c1*2 + part];
	    }

    printfQuda("Vector qkxTM packed on gpu form\n");
    packVector_flag = true;
  }
  else{
    errorQuda("Error not create host pointer");
  }

}

void QKXTM_Vector::loadVector(){

  if(packVector_flag == true && loadVector_flag == false){
    hipError_t error;
    if( (init_host_alloc == true) && (init_device_alloc == true) ){
      error = hipMemcpy(d_elem,h_elem,(bytes_total_length - bytes_ghost_length), hipMemcpyHostToDevice );
      if(error != hipSuccess) errorQuda("Error problem with load Vector on device");
      loadVector_flag = true;
      printfQuda("Vector qkxTM loaded on gpu\n");
    }
    else{
      errorQuda("Error try to load vector without allocate properly first");
    }
  }

}

void QKXTM_Vector::ghostToHost(){   // gpu collect ghost and send it to host

  // direction x ////////////////////////////////////
#ifdef MULTI_GPU
  if( localL[0] < totalL[0]){
    int position;
    int height = localL[1] * localL[2] * localL[3]; // number of blocks that we need
    size_t width = 2*sizeof(double);
    size_t spitch = localL[0]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;
  // set plus points to minus area
    position = (localL[0]-1)*localL[1]*localL[2]*localL[3];
    
      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[0]*nSpin*nColor*2 + mu*nColor*surface[0]*2 + c1*surface[0]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}
  // set minus points to plus area
    position = 0;

      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[0]*nSpin*nColor*2 + mu*nColor*surface[0]*2 + c1*surface[0]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}


  }
  // direction y ///////////////////////////////////

  if( localL[1] < totalL[1]){

    int position;
    int height = localL[2] * localL[3]; // number of blocks that we need
    size_t width = localL[0]*2*sizeof(double);
    size_t spitch = localL[1]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;

  // set plus points to minus area
    position = localL[0]*(localL[1]-1)*localL[2]*localL[3];
      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[1]*nSpin*nColor*2 + mu*nColor*surface[1]*2 + c1*surface[1]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}

  // set minus points to plus area
    position = 0;

      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[1]*nSpin*nColor*2 + mu*nColor*surface[1]*2 + c1*surface[1]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}

  }
  
  // direction z //////////////////////////////////
  if( localL[2] < totalL[2]){

    int position;
    int height = localL[3]; // number of blocks that we need
    size_t width = localL[1]*localL[0]*2*sizeof(double);
    size_t spitch = localL[2]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;

  // set plus points to minus area
    position = localL[0]*localL[1]*(localL[2]-1)*localL[3];
      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + minusGhost[2]*nSpin*nColor*2 + mu*nColor*surface[2]*2 + c1*surface[2]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}

  // set minus points to plus area
    position = 0;
      for(int mu = 0 ; mu < nSpin ; mu++)
	for(int c1 = 0 ; c1 < nColor ; c1++){
	  d_elem_offset = d_elem + mu*nColor*localVolume*2 + c1*localVolume*2 + position*2;  
	  h_elem_offset = h_elem + plusGhost[2]*nSpin*nColor*2 + mu*nColor*surface[2]*2 + c1*surface[2]*2;
	  hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
	}

  }


  // direction t /////////////////////////////////////
  if( localL[3] < totalL[3]){
    int position;
    int height = nSpin*nColor;
    size_t width = localL[2]*localL[1]*localL[0]*2*sizeof(double);
    size_t spitch = localL[3]*width;
    size_t dpitch = width;
    double *h_elem_offset = NULL;
    double *d_elem_offset = NULL;
  // set plus points to minus area
    position = localL[0]*localL[1]*localL[2]*(localL[3]-1);
    d_elem_offset = d_elem + position*2;
    h_elem_offset = h_elem + minusGhost[3]*nSpin*nColor*2;
    hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
  // set minus points to plus area
    position = 0;
    d_elem_offset = d_elem + position*2;
    h_elem_offset = h_elem + plusGhost[3]*nSpin*nColor*2;
    hipMemcpy2D(h_elem_offset,dpitch,d_elem_offset,spitch,width,height,hipMemcpyDeviceToHost);
  }


#endif
}

void QKXTM_Vector::cpuExchangeGhost(){ // cpus exchange links

#ifdef MULTI_GPU
  if(comm_size() > 1){

    MPI_Request request_recv[2*nDim];
    MPI_Request request_send[2*nDim];
    int back_nbr[4] = {X_BACK_NBR,Y_BACK_NBR,Z_BACK_NBR,T_BACK_NBR};             
    int fwd_nbr[4] = {X_FWD_NBR,Y_FWD_NBR,Z_FWD_NBR,T_FWD_NBR};

    // direction x
    if(localL[0] < totalL[0]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[0]*nSpin*nColor*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[0]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + minusGhost[0]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[0], 0, &(request_recv[0]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[0], 0, &(request_send[0]));
      comm_wait(&(request_recv[0])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[0]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + plusGhost[0]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[0], 1, &(request_recv[1]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[0], 1, &(request_send[1]));
      comm_wait(&(request_recv[1])); // blocking until receive finish
      
    }
    // direction y
    if(localL[1] < totalL[1]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[1]*nSpin*nColor*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[1]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + minusGhost[1]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[1], 2, &(request_recv[2]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[1], 2, &(request_send[2]));
      comm_wait(&(request_recv[2])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[1]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + plusGhost[1]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[1], 3, &(request_recv[3]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[1], 3, &(request_send[3]));
      comm_wait(&(request_recv[3])); // blocking until receive finish
    }

    // direction z
    if(localL[2] < totalL[2]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[2]*nSpin*nColor*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[2]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + minusGhost[2]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[2], 4, &(request_recv[4]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[2], 4, &(request_send[4]));
      comm_wait(&(request_recv[4])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[2]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + plusGhost[2]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[2], 5, &(request_recv[5]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[2], 5, &(request_send[5]));
      comm_wait(&(request_recv[5])); // blocking until receive finish
    }


    // direction t
    if(localL[3] < totalL[3]){
      double *pointer_receive = NULL;
      double *pointer_send = NULL;
      size_t nbytes = surface[3]*nSpin*nColor*2*sizeof(double);

      // send to plus
      pointer_receive = h_ext_ghost + (minusGhost[3]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + minusGhost[3]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, back_nbr[3], 6, &(request_recv[6]));
      comm_send_with_tag(pointer_send, nbytes, fwd_nbr[3], 6, &(request_send[6]));
      comm_wait(&(request_recv[6])); // blocking until receive finish

      // send to minus
      pointer_receive = h_ext_ghost + (plusGhost[3]-localVolume)*nSpin*nColor*2;
      pointer_send = h_elem + plusGhost[3]*nSpin*nColor*2;
      comm_recv_with_tag(pointer_receive, nbytes, fwd_nbr[3], 7, &(request_recv[7]));
      comm_send_with_tag(pointer_send, nbytes, back_nbr[3], 7, &(request_send[7]));
      comm_wait(&(request_recv[7])); // blocking until receive finish
    }


  }
#endif

}

void QKXTM_Vector::ghostToDevice(){ // simple cudamemcpy to send ghost to device
#ifdef MULTI_GPU
  if(comm_size() > 1){
    double *host = h_ext_ghost;
    double *device = d_elem + localVolume*nSpin*nColor*2;
    hipMemcpy(device,host,bytes_ghost_length,hipMemcpyHostToDevice);
  }
#endif
}

void QKXTM_Vector::bindVector(){
  hipError_t error;
  if( vector_binded == false ){
    error = hipBindTexture(0,vectorTex,d_elem,bytes_total_length);
    if(error != hipSuccess) errorQuda("Problem bind Texture");
  }
  vector_binded = true;
}

void QKXTM_Vector::unbindVector(){
  hipError_t error;
  if(vector_binded == true){
    error = hipUnbindTexture(vectorTex);
    if(error != hipSuccess) errorQuda("Problem unbind Texture");
  }
  vector_binded = false;
}


////////////////////////////////////////////////////////////////// for prop ///////////////////////////////////////////////
